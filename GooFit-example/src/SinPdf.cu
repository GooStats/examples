#include "SinPdf.h"

EXEC_TARGET fptype device_Sin (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x = evt[indices[2 + indices[0]]]; 
  fptype alpha = p[indices[1]];
  fptype beta = p[indices[2]];
  const int cIndex = RO_CACHE(indices[3]);
  const fptype drift = RO_CACHE(functorConstants[cIndex]); // ok

  fptype ret = SIN(alpha*x+beta)+1+drift; 
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_Sin = device_Sin; 

__host__ SinPdf::SinPdf (std::string n, Variable* _x, Variable* alpha, Variable* beta,fptype drift) 
  : GooPdf(_x, n) 
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(alpha));
  pindices.push_back(registerParameter(beta));
  pindices.push_back(registerConstants(1)); // 5
  fptype hostConstants[1] = { drift };
  MEMCPY_TO_SYMBOL(functorConstants, hostConstants, sizeof(fptype), cIndex*sizeof(fptype), hipMemcpyHostToDevice); 
  GET_FUNCTION_ADDR(ptr_to_Sin);
  initialise(pindices); 
}


__host__ fptype SinPdf::integrate (fptype lo, fptype hi) const {
  fptype alpha = host_params[host_indices[parameters + 1]]; 
  fptype beta = host_params[host_indices[parameters + 2]]; 
  if(alpha==0) return (SIN(beta)+1)*(hi-lo);

  fptype ret = (COS(alpha*lo+beta) - COS(alpha*hi+beta))/alpha+hi-lo;
  return ret; 
}

